#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#define INPUT_FILE_PATH "C:\\Users\\User\\source\\repos\\cudaSLAERelaxationMethod\\test_data.txt"
#define OUTPUT_FILE_PATH "output.txt"

void printMassive(double* mas, int size);
void printMatrix(double** matrix, int size_x, int size_y);

/*
Функция ядра для приведения исходных матриц коэффициентов и свободных членов
    задачи СЛАУ с ленточной структурой данных к требуемому в методе реалксации виду
Работает с одномерной сеткой из одного двумерного блока с рекомндумыми размерами:
    Высота: количество уравнений n; Ширина: ширина ленты
Принимает матрицу коэффициентов A,
          матрица-столбец свободных членов B,
          порядок матрицы коэффициентов n,
          матрицу преобразованных коэффициентов P,
          матрицу преобразованных свободных членов C
*/
__global__ void 
relaxationMatrixReductionKernel(double** A, double* B, int n, double** P, double* C) {
    // идентификтаоры блока и потока
    /*int bx = blockIdx.x;
    int by = blockIdx.y;*/
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    //размеры блока по x и по y
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    //число потоков в блоке
    int tnum = bdx * bdy;

    //вычисление приведённой матрицы коэффициентов
    for (int ptrx = tx; ptrx < n; ptrx += bdx) {
        for (int ptry = ty; ptry < n; ptry += bdy) {
            P[ptrx][ptry] = -A[ptrx][ptry] / A[ptrx][ptrx];
        }
    }

    //вычисление приведённой матрицы-столбца
    for (int tind = tx + ty * bdx; tind < n; tind += tnum) {
        C[tind] = B[tind] / A[tind][tind];
    }
}

/*
Функция ядра для решения СЛАУ с ленточной сруктурой матрицы
Работает с одномерной сеткой из одного двумерного блока с рекомндумыми размерами:
    Высота: количество уравнений n; Ширина: ширина ленты
Принимает матрицу коэффициентов A,
          массив свободных членов B,
          массив ответов X, куда записываются найденные значения неизвестных,
          приближения eps.
*/
//__global__ void
//relaxationIterationKernel(double** P, double* C, double* X, double eps)
//{
//    // идентификтаоры блока и потока
//    /*int bx = blockIdx.x;
//    int by = blockIdx.y;*/
//    int tx = threadIdx.x;
//    int ty = threadIdx.y;
//
//
//
//}

/*

*/
double* relaxationMethod(double** A, double* B, int n ) {
    double** ADev;
    double* BDev;
    //float* nDev;
    double** PDev;
    double* CDev;
    hipMalloc(&ADev, n * n * sizeof(double));
    hipMalloc(&BDev, n * sizeof(double));
    //hipMalloc(&nDev, sizeof(int));
    hipMalloc(&PDev, n * n * sizeof(double));
    hipMalloc(&CDev, n * sizeof(double));

    hipMemcpy(ADev, A, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(BDev, B, n * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(nDev, n, sizeof(int), hipMemcpyHostToDevice);
    relaxationMatrixReductionKernel <<<1, dim3(n, n)>>>(ADev, BDev, n, PDev CDev);

    double** P;
    double* C;

    hipMemcpy(P, PDev, n * n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(C, CDev, n * sizeof(double), hipMemcpyDeviceToHost);

    printMatrix(P, n, n);
    printf("\n");
    printMassive(C, n);

    hipFree(ADev);
    hipFree(BDev);
    //hipFree(nDev);

    
}

double** readMatrix(FILE *input, int size_x, int size_y) {
    double** matrix = new double*[size_x];
    for (int i = 0; i < size_x; i++) {
        matrix[i] = new double[size_y];
        for (int j = 0; j < size_y; j++) {
            fscanf(input, "%lf", &matrix[i][j]);
        }
    }
    return matrix;
}

double* readMassive(FILE* input, int size) {
    return readMatrix(input,  1, size)[0];
}

void printMassive(double* mas, int size) {
    for (int i = 0; i < size; i++) {
        printf("%lf ", mas[i]);
    }
    printf("\n");
}
    
void printMatrix(double** matrix, int size_x, int size_y) {
    for (int i = 0; i < size_x; i++) {
        printMassive(matrix[i], size_y);
    }
}

int main(void)
{

    FILE* input_data;
    if ((input_data = fopen(INPUT_FILE_PATH, "r")) == NULL)
    {
        printf("Input file open error");
        return 0;
    }

    FILE* output_data;
    if ((output_data = fopen(OUTPUT_FILE_PATH, "w")) == NULL)
    {
        printf("Output file open error");
        return 0;
    }

    int n = 0;
    fscanf(input_data, "%d", &n);
    double** A = readMatrix(input_data, n, n);
    double* B = readMassive(input_data, n);

    printMatrix(A, n, n); 
    printf("\n");
    printMassive(B, n);

    return 0;
}