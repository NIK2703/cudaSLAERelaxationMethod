﻿
#include <hip/hip_runtime.h>
#include <stdio.h>

#define INPUT_FILE_PATH "C:\\Users\\User\\source\\repos\\cudaSLAERelaxationMethod\\test_data.txt"
#define OUTPUT_FILE_PATH "output.txt"

void printMassive(double* mas, int size);
void printMatrix(double** matrix, int size_x, int size_y);

/*
Функция ядра для приведения исходных матриц коэффициентов и свободных членов
    задачи СЛАУ с ленточной структурой данных к требуемому в методе реалксации виду
Работает с одномерной сеткой из одного двумерного блока с рекомндумыми размерами:
    Высота: количество уравнений n; Ширина: ширина ленты
Принимает матрицу коэффициентов A,
          матрица-столбец свободных членов B,
          порядок матрицы коэффициентов n,
          матрицу преобразованных коэффициентов P,
          матрицу преобразованных свободных членов C
*/
__global__ void
relaxationMatrixReductionKernel(double* A, double* B, int n, double* P, double* C) {
    // идентификтаоры блока и потока
    /*int bx = blockIdx.x;
    int by = blockIdx.y;*/
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    //размеры блока по x и по y
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    //число потоков в блоке
    int tnum = bdx * bdy;

    //вычисление приведённой матрицы коэффициентов
   for (int ptrx = tx; ptrx < n; ptrx += bdx) {
        for (int ptry = ty; ptry < n; ptry += bdy) {
            P[ptrx + ptry * bdx] = -A[ptrx + ptry * bdx] / A[ptrx + ptrx * bdx];
        }
    }

    //вычисление приведённой матрицы-столбца
    for (int tind = tx + ty * bdx; tind < n; tind += tnum) {
        C[tind] = B[tind] / A[tind + tind * bdx];
    }
}

/*
Функция ядра для решения СЛАУ с ленточной сруктурой матрицы
Работает с одномерной сеткой из одного двумерного блока с рекомндумыми размерами:
    Высота: количество уравнений n; Ширина: ширина ленты
Принимает матрицу коэффициентов A,
          массив свободных членов B,
          массив ответов X, куда записываются найденные значения неизвестных,
          приближения eps.
*/
//__global__ void
//relaxationIterationKernel(double** P, double* C, double* X, double eps)
//{
//    // идентификтаоры блока и потока
//    /*int bx = blockIdx.x;
//    int by = blockIdx.y;*/
//    int tx = threadIdx.x;
//    int ty = threadIdx.y;
//
//
//
//}

//double* stretchMatrix(double** matrix, int size_x, int size_y) {
//    double* stretchedMatrix = new double[size_x * size_y];
//    for (int i = 0; i < size_x; i++) {
//        for (int j = 0; j < size_y; j++) {
//            stretchedMatrix[i + j * size_x] = matrix[i][j];
//        }
//    }
//    return stretchedMatrix;
//}
//
//double** squeezeMatrix(double* matrix, int size_x, int size_y) {
//    double** squeezedMatrix = new double*[size_x];
//    for (int i = 0; i < size_x; i++) {
//        squeezedMatrix[i] = new double[size_y];
//        for (int j = 0; j < size_y; j++) {
//            squeezedMatrix[i][j] = matrix[i + j * size_x] ;
//        }
//    }
//    return squeezedMatrix;
//}

double* stretchMatrix(double** matrix, int size_x, int size_y) {
    double* stretchedMatrix = new double[size_x * size_y];
    for (int i = 0; i < size_x; i++) {
        for (int j = 0; j < size_y; j++) {
            stretchedMatrix[i * size_y + j] = matrix[i][j];
        }
    }
    return stretchedMatrix;
}

double** squeezeMatrix(double* matrix, int size_x, int size_y) {
    double** squeezedMatrix = new double* [size_x];
    for (int i = 0; i < size_x; i++) {
        squeezedMatrix[i] = new double[size_y];
        for (int j = 0; j < size_y; j++) {
            squeezedMatrix[i][j] = matrix[i * size_y + j];
        }
    }
    return squeezedMatrix;
}

/*

*/
double* relaxationMethod(double** A, double* B, int n ) {
    double* ADev;
    double* BDev;
    //float* nDev;
    double* PDev;
    double* CDev;

    double* stretchedA = stretchMatrix(A, n, n);

    hipMalloc(&ADev, n * n * sizeof(double));
    hipMalloc(&BDev, n * sizeof(double));
    //cudaMalloc(&nDev, sizeof(int));
    hipMalloc(&PDev, n * n * sizeof(double));
    hipMalloc(&CDev, n * sizeof(double));

    hipMemcpy(ADev, stretchedA, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(BDev, B, n * sizeof(double), hipMemcpyHostToDevice);
    //cudaMemcpy(nDev, n, sizeof(int), cudaMemcpyHostToDevice);

    relaxationMatrixReductionKernel <<<1, dim3(n, n)>>>(ADev, BDev, n, PDev, CDev);

    double* stretchedP = new double[n * n];
    double* C = new double[n];

    hipMemcpy(stretchedP, PDev, n * n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(C, CDev, n * sizeof(double), hipMemcpyDeviceToHost);

    double** P = squeezeMatrix(stretchedP, n, n);

    printMatrix(P, n, n);
    printf("\n");
    printMassive(C, n);

    hipFree(ADev);
    hipFree(BDev);
    //cudaFree(nDev);

    return NULL;
}

double** readMatrix(FILE *input, int size_x, int size_y) {
    double** matrix = new double*[size_x];
    for (int i = 0; i < size_x; i++) {
        matrix[i] = new double[size_y];
        for (int j = 0; j < size_y; j++) {
            fscanf(input, "%lf", &matrix[i][j]);
        }
    }
    return matrix;
}

double* readMassive(FILE* input, int size) {
    return readMatrix(input,  1, size)[0];
}

void printMassive(double* mas, int size) {
    for (int i = 0; i < size; i++) {
        printf("%lf ", mas[i]);
    }
    printf("\n");
}
    
void printMatrix(double** matrix, int size_x, int size_y) {
    for (int i = 0; i < size_x; i++) {
        printMassive(matrix[i], size_y);
    }
}

int main(void)
{

    FILE* input_data;
    if ((input_data = fopen(INPUT_FILE_PATH, "r")) == NULL)
    {
        printf("Input file open error");
        return 0;
    }

    FILE* output_data;
    if ((output_data = fopen(OUTPUT_FILE_PATH, "w")) == NULL)
    {
        printf("Output file open error");
        return 0;
    }

    int n = 0;
    fscanf(input_data, "%d", &n);
    double** A = readMatrix(input_data, n, n);
    double* B = readMassive(input_data, n);

    relaxationMethod(A, B, n);

    /*printMatrix(A, n, n);
    printf("\n");

    double* stretchedA = stretchMatrix(A, n, n);
    printMassive(stretchedA, n * n);
    printf("\n");

    double** squeezedA = squeezeMatrix(stretchedA, n, n);
    printMatrix(squeezedA, n, n);*/

    

   /* printMatrix(A, n, n); 
    printf("\n");
    printMassive(B, n);*/

    return 0;
}